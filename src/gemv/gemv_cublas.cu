// Matrix-vector Multiplication using cuBLAS
// (C) 2022 Bagus Hanindhito

#include <CUDA_Bench/gemv/gemv_cublas.cuh>
#include <CUDA_Bench/gemv/gemv_cublas_launch_fp.cuh>
#include <CUDA_Bench/gemv/gemv_cublas_launch_int.cuh>
#include <CUDA_Bench/gemv/gemv_util.cuh>
#include <CUDA_Bench/gemv/gemv_global.cuh>
#include <CUDA_Bench/util/gpuinfo.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <CUDA_Bench/util/precision_select.cuh>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include <nvbench/nvbench.cuh>

int gemv_cublas()
{
    // Detect Available CUDA Devices
    int nDevices;
    gpuErrchk(hipGetDeviceCount(&nDevices));
    print_cuda_device_info(nDevices);
    if(nDevices>0) {std::cout << "[WARN] This program does not currently support Multi-GPU run.\n";}

    // Call cuBLAS Launcher
    if      (gmulprecision==PRECISION_FP64 && gaccprecision==PRECISION_FP64)
    {
        if(gprofiling) {NVBENCH_BENCH(gemv_cublas_launch_fp_double_double_double); NVBENCH_MAIN_BODY(gargc_nvbench, gargv_nvbench);}
        else{gemv_cublas_launch_fp_double_double_double();}
    }
    else if (gmulprecision==PRECISION_FP32 && gaccprecision==PRECISION_FP32)
    {
        if(gprofiling) {NVBENCH_BENCH(gemv_cublas_launch_fp_float_float_float); NVBENCH_MAIN_BODY(gargc_nvbench, gargv_nvbench);}
        else{gemv_cublas_launch_fp_float_float_float();}
    }
    else if ((gmulprecision==PRECISION_FP16) && gaccprecision==PRECISION_FP32)
    {
        if(gprofiling) {NVBENCH_BENCH(gemv_cublas_launch_fp_float_half_float); NVBENCH_MAIN_BODY(gargc_nvbench, gargv_nvbench);}
        else{gemv_cublas_launch_fp_float_half_float();}
    }
    else if (gmulprecision==PRECISION_FP16 && gaccprecision==PRECISION_FP16)
    {
        if(gprofiling) {NVBENCH_BENCH(gemv_cublas_launch_fp_half_half_half); NVBENCH_MAIN_BODY(gargc_nvbench, gargv_nvbench);}
        else{gemv_cublas_launch_fp_half_half_half();}
    }
    else if (gmulprecision==PRECISION_INT8 && gaccprecision==PRECISION_INT8)
    {
        if(gprofiling) {NVBENCH_BENCH(gemv_cublas_launch_fp_int8_int8_int8); NVBENCH_MAIN_BODY(gargc_nvbench, gargv_nvbench);}
        else{gemv_cublas_launch_fp_int8_int8_int8();}
    }
    else
    {
        std::cerr <<"[ERR!] Precision combination is not supported\n\n\n";
        std::exit(1);
    } 
    return 0;
}