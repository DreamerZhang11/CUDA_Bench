// Matrix-matrix Multiplication using cuBLAS
// (C) 2022 Bagus Hanindhito

#include <CUDA_Bench/gemm/gemm_cublas.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int gemm_cublas(int dim_M, int dim_N, int dim_K)
{
    hipblasStatus_t stat;    // CUBLAS functions status
	hipblasHandle_t handle;	// CUBLAS context

    gpuErrchk(hipblasCreate(&handle));

}