#include <CUDA_Bench/util/precision_select.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <CUDA_Bench/gemm/gemm_util.cuh>
#include <CUDA_Bench/gemm/gemm_global.cuh>
#include <CUDA_Bench/gemm/gemm_cublas_launch_fp.cuh>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <nvbench/nvbench.cuh>

int gemm_cublas_launch_fp_double_double_double()
{
    gemm_cublas_launch_fp<double, double, double>();
    return 0;
}

int gemm_cublas_launch_fp_double_double_double(nvbench::state& state)
{
    gemm_cublas_launch_fp<double, double, double>(state);
    return 0;
}

int gemm_cublas_launch_fp_float_float_float()
{
    gemm_cublas_launch_fp<float, float, float>();
    return 0;
}

int gemm_cublas_launch_fp_float_float_float(nvbench::state& state)
{
    gemm_cublas_launch_fp<float, float, float>(state);
    return 0;
}

int gemm_cublas_launch_fp_float_half_float()
{
    gemm_cublas_launch_fp<float, half, float>();
    return 0;
}

int gemm_cublas_launch_fp_float_half_float(nvbench::state& state)
{
    gemm_cublas_launch_fp<float, half, float>(state);
    return 0;
}

int gemm_cublas_launch_fp_half_half_half()
{
    gemm_cublas_launch_fp<half, half, half>();
    return 0;
}

int gemm_cublas_launch_fp_half_half_half(nvbench::state& state)
{
    gemm_cublas_launch_fp<half, half, half>(state);
    return 0;
}