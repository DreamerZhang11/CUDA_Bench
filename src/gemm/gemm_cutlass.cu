#include "hip/hip_runtime.h"
// Matrix-matrix Multiplication using cutlass
// (C) 2022 Bagus Hanindhito

#include <CUDA_Bench/gemm/gemm_cutlass.cuh>
#include <CUDA_Bench/gemm/gemm_cutlass_launch_int.cuh>
#include <CUDA_Bench/util/gpuinfo.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <CUDA_Bench/util/precision_select.cuh>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>

int gemm_cutlass(int dim_M, int dim_N, int dim_K, Precision mulprecision, Precision accprecision, int num_iter, bool print_result, bool tensor_cores, bool profiling)
{
     // Detect Available CUDA Devices
    int nDevices;
    gpuErrchk(hipGetDeviceCount(&nDevices));
    print_cuda_device_info(nDevices);
    if(nDevices>0) {std::cout << "[WARN] This program does not currently support Multi-GPU run.\n";}

    // Detect Device Capability
    hipDeviceProp_t props;
    gpuErrchk(hipGetDeviceProperties(&props, 0));
    GPUARCH gpuarch;

    switch(props.major * 10 + props.minor)
    {
        case 70: {gpuarch=GPUARCH_VOLTA;  break;}
        case 75: {gpuarch=GPUARCH_TURING; break;}
        case 80: {gpuarch=GPUARCH_AMPERE; break;}
        case 86: {gpuarch=GPUARCH_AMPERE; break;}
        default: {gpuarch=GPUARCH_OTHER; break;}
    }


    // Precision Compability Check
    if(mulprecision==PRECISION_INT8 && accprecision==PRECISION_INT8)
    {
        std::cout << "[WARN] Promoting accumulation precision to int32 to maintain compability\n";
        if(tensor_cores)
        {
            switch(gpuarch)
            {
                case GPUARCH_VOLTA : {std::cout << "[ERR!] Volta Tensor Cores do not support int8. Please use CUDA Cores instead\n"; exit(1); break;}
                case GPUARCH_TURING: {gemm_cutlass_launch_turing_int32_int8_int32_tc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                case GPUARCH_AMPERE: {gemm_cutlass_launch_ampere_int32_int8_int32_tc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                default: {std::cout << "[ERR!] GPU Compute Capability is lower than it is required\n"; exit(1); break;}
            }
            
        }
        else
        {
            switch(gpuarch)
            {
                case GPUARCH_VOLTA : {gemm_cutlass_launch_volta_int32_int8_int32_ntc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                case GPUARCH_TURING: {gemm_cutlass_launch_turing_int32_int8_int32_ntc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                case GPUARCH_AMPERE: {gemm_cutlass_launch_ampere_int32_int8_int32_ntc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                default: {std::cout << "[ERR!] GPU Compute Capability is lower than it is required\n"; exit(1); break;}
            }
        }
    }
    else if(mulprecision==PRECISION_INT4 && (accprecision==PRECISION_INT8 || accprecision==PRECISION_INT4))
    {
        std::cout << "[WARN] Promoting accumulation precision to int32 to maintain compability\n";
        if(tensor_cores)
        {
            switch(gpuarch)
            {
                case GPUARCH_VOLTA : {std::cout << "[ERR!] Volta Tensor Cores do not support int4\n"; exit(1); break;}
                case GPUARCH_TURING: {gemm_cutlass_launch_turing_int32_int4_int32_tc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                case GPUARCH_AMPERE: {gemm_cutlass_launch_ampere_int32_int4_int32_tc(dim_M, dim_N, dim_K, num_iter, print_result, profiling); break;}
                default: {std::cout << "[ERR!] GPU Compute Capability is lower than it is required\n"; exit(1); break;}
            }
            
        }
        else
        {
            std::cout << "[ERR!] Operations involving int4 requires the use of Tensor Cores\n"; 
            exit(1);
        }
    }
    return 0;
}