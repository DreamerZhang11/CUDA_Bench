#include <CUDA_Bench/util/precision_select.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <CUDA_Bench/fir/fir_util.cuh>
#include <CUDA_Bench/fir/fir_global.cuh>
#include <CUDA_Bench/fir/fir_cublas_launch_fp.cuh>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <nvbench/nvbench.cuh>

int fir_cublas_launch_fp_double_double_double()
{
    fir_cublas_launch_fp<double, double, double>();
    return 0;
}

int fir_cublas_launch_fp_double_double_double(nvbench::state& state)
{
    fir_cublas_launch_fp<double, double, double>(state);
    return 0;
}

int fir_cublas_launch_fp_float_float_float()
{
    fir_cublas_launch_fp<float, float, float>();
    return 0;
}

int fir_cublas_launch_fp_float_float_float(nvbench::state& state)
{
    fir_cublas_launch_fp<float, float, float>(state);
    return 0;
}

int fir_cublas_launch_fp_float_half_float()
{
    fir_cublas_launch_fp<float, half, float>();
    return 0;
}

int fir_cublas_launch_fp_float_half_float(nvbench::state& state)
{
    fir_cublas_launch_fp<float, half, float>(state);
    return 0;
}

int fir_cublas_launch_fp_half_half_half()
{
    fir_cublas_launch_fp<half, half, half>();
    return 0;
}

int fir_cublas_launch_fp_half_half_half(nvbench::state& state)
{
    fir_cublas_launch_fp<half, half, half>(state);
    return 0;
}