
#include <CUDA_Bench/util/precision_select.cuh>
#include <CUDA_Bench/util/gpucheck.cuh>
#include <CUDA_Bench/gemm/gemm_util.cuh>
#include <CUDA_Bench/gemm/gemm_global.cuh>
#include <CUDA_Bench/gemm/gemm_cublas_launch_fp.cuh>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>